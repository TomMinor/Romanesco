#include "hip/hip_runtime.h"
// pos 0.299997 0 1.5747e-05
// rot 0 -1.5708 0
// fov 60


#include "romanescocore.h"
#include "tunneltest.h"

HIT_PROGRAM float hit(float3 x, uint maxIterations, float global_t)
{
	TunnelTest sdf(maxIterations);
	sdf.evalParameters();
    	sdf.setTime(global_t);
    	sdf.x -= global_t;
    	sdf.setTranslateHook( 0, x );

	return sdf.evalDistance(x);
}
