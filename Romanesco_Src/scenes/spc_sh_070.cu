#include "hip/hip_runtime.h"
// pos 3.525 0 7.33048e-06
// rot 0 1.5707 0
// fov 60


#include "romanescocore.h"
#include "tunneltest.h"

HIT_PROGRAM float4 hit(float3 x, int maxIterations, float global_t)
{
	TunnelTest sdf(maxIterations);
	sdf.evalParameters();
    	sdf.setTime(global_t);
    	sdf.setTranslateHook( 0, x - make_float3(-global_t, 0.0f, 0.0f) );

	return make_float4( sdf.evalDistance(x), 
						sdf.getTrap0(), 
						sdf.getTrap1(), 
						sdf.getTrap2() );
}
