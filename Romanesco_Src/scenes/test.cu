#include "hip/hip_runtime.h"
// pos 3.075 0 5.70148e-06
// rot 0 1.5708 0
// fov 60

#include "romanescocore.h"

HIT_PROGRAM float hit(float3 x, uint maxIterations, float global_t)
{
	Mandelbulb sdf(maxIterations);
	sdf.evalParameters();
    	sdf.setTime(global_t);

	return sdf.evalDistance(x);
}
